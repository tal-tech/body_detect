#include "hip/hip_runtime.h"
#include "caffe/util/bbox_util.hpp"
#include "caffe/core/common.hpp"


namespace facethink {

  template <typename Dtype>
  __global__ void PermuteDataKernel(const int nthreads,
				    const Dtype* data, const int num_classes, const int num_data,
				    const int num_dim, Dtype* new_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      const int i = index % num_dim;
      const int c = (index / num_dim) % num_classes;
      const int d = (index / num_dim / num_classes) % num_data;
      const int n = index / num_dim / num_classes / num_data;
      const int new_index = ((n * num_classes + c) * num_data + d) * num_dim + i;
      new_data[new_index] = data[index];
    }
  }

  template <typename Dtype>
  void PermuteDataGPU(const int nthreads,
		      const Dtype* data, const int num_classes, const int num_data,
		      const int num_dim, Dtype* new_data) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteDataKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, data, num_classes, num_data,
				num_dim, new_data);
    CUDA_POST_KERNEL_CHECK;
  }

  template void PermuteDataGPU(const int nthreads,
			       const float* data, const int num_classes, const int num_data,
			       const int num_dim, float* new_data);


  ///////////////////////////////////////////////////////////////////////////////////////

  template <typename Dtype>
  __global__ void DecodeBBoxesKernel(const int nthreads,
				     const Dtype* loc_data, const Dtype* prior_data,
				     const CodeType code_type, const bool variance_encoded_in_target,
				     const int num_priors, const bool share_location,
				     const int num_loc_classes, const int background_label_id,
				     const bool clip_bbox, Dtype* bbox_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      const int i = index % 4;
      const int c = (index / 4) % num_loc_classes;
      const int d = (index / 4 / num_loc_classes) % num_priors;
      if (!share_location && c == background_label_id) {
	// Ignore background class if not share_location.
	return;
      }
      const int pi = d * 4;
      const int vi = pi + num_priors * 4;
      if (code_type == CORNER) {
	if (variance_encoded_in_target) {
	  // variance is encoded in target, we simply need to add the offset
	  // predictions.
	  bbox_data[index] = prior_data[pi + i] + loc_data[index];
	} else {
	  // variance is encoded in bbox, we need to scale the offset accordingly.
	  bbox_data[index] =
	    prior_data[pi + i] + loc_data[index] * prior_data[vi + i];
	}
      } else if (code_type == CENTER_SIZE) {
	const Dtype p_xmin = prior_data[pi];
	const Dtype p_ymin = prior_data[pi + 1];
	const Dtype p_xmax = prior_data[pi + 2];
	const Dtype p_ymax = prior_data[pi + 3];
	const Dtype prior_width = p_xmax - p_xmin;
	const Dtype prior_height = p_ymax - p_ymin;
	const Dtype prior_center_x = (p_xmin + p_xmax) / 2.;
	const Dtype prior_center_y = (p_ymin + p_ymax) / 2.;

	const Dtype xmin = loc_data[index - i];
	const Dtype ymin = loc_data[index - i + 1];
	const Dtype xmax = loc_data[index - i + 2];
	const Dtype ymax = loc_data[index - i + 3];

	Dtype decode_bbox_center_x, decode_bbox_center_y;
	Dtype decode_bbox_width, decode_bbox_height;
	if (variance_encoded_in_target) {
	  // variance is encoded in target, we simply need to retore the offset
	  // predictions.
	  decode_bbox_center_x = xmin * prior_width + prior_center_x;
	  decode_bbox_center_y = ymin * prior_height + prior_center_y;
	  decode_bbox_width = exp(xmax) * prior_width;
	  decode_bbox_height = exp(ymax) * prior_height;
	} else {
	  // variance is encoded in bbox, we need to scale the offset accordingly.
	  decode_bbox_center_x =
	    prior_data[vi] * xmin * prior_width + prior_center_x;
	  decode_bbox_center_y =
	    prior_data[vi + 1] * ymin * prior_height + prior_center_y;
	  decode_bbox_width =
	    exp(prior_data[vi + 2] * xmax) * prior_width;
	  decode_bbox_height =
	    exp(prior_data[vi + 3] * ymax) * prior_height;
	}

	switch (i) {
        case 0:
          bbox_data[index] = decode_bbox_center_x - decode_bbox_width / 2.;
          break;
        case 1:
          bbox_data[index] = decode_bbox_center_y - decode_bbox_height / 2.;
          break;
        case 2:
          bbox_data[index] = decode_bbox_center_x + decode_bbox_width / 2.;
          break;
        case 3:
          bbox_data[index] = decode_bbox_center_y + decode_bbox_height / 2.;
          break;
	}
      } else if (code_type == CORNER_SIZE) {
	const Dtype p_xmin = prior_data[pi];
	const Dtype p_ymin = prior_data[pi + 1];
	const Dtype p_xmax = prior_data[pi + 2];
	const Dtype p_ymax = prior_data[pi + 3];
	const Dtype prior_width = p_xmax - p_xmin;
	const Dtype prior_height = p_ymax - p_ymin;
	Dtype p_size;
	if (i == 0 || i == 2) {
	  p_size = prior_width;
	} else {
	  p_size = prior_height;
	}
	if (variance_encoded_in_target) {
	  // variance is encoded in target, we simply need to add the offset
	  // predictions.
	  bbox_data[index] = prior_data[pi + i] + loc_data[index] * p_size;
	} else {
	  // variance is encoded in bbox, we need to scale the offset accordingly.
	  bbox_data[index] =
	    prior_data[pi + i] + loc_data[index] * prior_data[vi + i] * p_size;
	}
      } else {
	// Unknown code type.
      }
      if (clip_bbox) {
	bbox_data[index] = max(min(bbox_data[index], Dtype(1.)), Dtype(0.));
      }
    }
  }

  template <typename Dtype>
  void DecodeBBoxesGPU(const int nthreads,
		       const Dtype* loc_data, const Dtype* prior_data,
		       const CodeType code_type, const bool variance_encoded_in_target,
		       const int num_priors, const bool share_location,
		       const int num_loc_classes, const int background_label_id,
		       const bool clip_bbox, Dtype* bbox_data) {
    // NOLINT_NEXT_LINE(whitespace/operators)
    DecodeBBoxesKernel<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, loc_data, prior_data, code_type,
				variance_encoded_in_target, num_priors, share_location, num_loc_classes,
				background_label_id, clip_bbox, bbox_data);
    CUDA_POST_KERNEL_CHECK;
  }

  template void DecodeBBoxesGPU(const int nthreads,
				const float* loc_data, const float* prior_data,
				const CodeType code_type, const bool variance_encoded_in_target,
				const int num_priors, const bool share_location,
				const int num_loc_classes, const int background_label_id,
				const bool clip_bbox, float* bbox_data);


} // namespace facethink
